#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
// Tomás Oliveira e Silva, December 2016
//
// ACA 2016/2017
//
// Reference implementation of the MD5 message digest for a single block of 512 bits (16 32-bit words) on
// * a CPU
// * optionaly, on a CUDA device, or
//
// The memory layout of the CUDA implementation is, on purpose, not optimal (*** OPTIMIZE IT ***)
//

//#define _GNU_SOURCE
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "common/common.h"
#include <hip/hip_runtime.h>
#include "md5.h"
#define max_device_memory  ((size_t)1.3e9) // no more than 1.3GB


//
// Measure real elapsed time
//

static double get_delta_time(void)
{
  static struct timespec t0,t1;

  t0 = t1;
  if(clock_gettime(CLOCK_MONOTONIC,&t1) != 0)
  {
    perror("clock_gettime");
    exit(1);
  }
  return (double)(t1.tv_sec - t0.tv_sec) + 1.0e-9 * (double)(t1.tv_nsec - t0.tv_nsec);
}


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
// CPU reference implementation
//

static void md5_cpu(const unsigned int data[16],unsigned int hash[4])
{
# define T(i)  K  // in our unmodified MD5 message digest the T[i] constants can all be stored in the same place
  unsigned int K; // place where the T[i] constants will be stored

  MD5(data,1,hash,1);
# undef T
}

static void md5_cpu_verification(void)
{
  static char text[56] = "ACA 2016/2017 (MD5 test)\n"; // text to hash
  unsigned int data[16],hash[4];
  char command[128];
  int i;

  //
  // build 512-bit block
  //
  for(i = 0;i < 16;i++)
    data[i] = 0;
  for(i = 0;i < 55 && text[i] != '\0';i++)
    data[i / 4] |= ((unsigned int)text[i] & 0xFF) << (8 * (i % 4));
  data[i / 4] |= 0x80 << (8 * (i % 4)); // append 0x80
  data[14] = 8 * i; // number of bits
  //
  // compute and display the MD5 message digest using our code
  //
  md5_cpu(data,hash);
  printf("MD5 test (the following two lines should be equal)\n");
  for(i = 0;i < 16;i++)
    printf("%02x",0xFF & (hash[i / 4] >> (8 * (i % 4)))); // least significant byte first!
  printf("  -\n");
  //
  // compute and display the MD5 message digest using the md5sum program
  //
  sprintf(command,"echo -n '%s' | md5sum",text);
  if(system(command) != 0)
    fprintf(stderr,"\"%s\" failed!\n",command);
  printf("\n");
}

static void md5_cpu_speed(int n_blocks,int n_passes)
{
  double dt1,dt2,dt,dt_sum,dt_min,dt_max;
  unsigned int data[16],hash[4];
  int i,n,pass;

  if(n_blocks <= 0)
    n_blocks = 1 << 20;
  if(n_blocks > (1 << 24))
    n_blocks = 1 << 24;
  if(n_passes <= 0)
    n_passes = 10;
  if(n_passes > 100)
    n_passes = 100;
  printf("md5_cpu_speed (%d block%s, %d+2 passes):\n",n_blocks,(n_blocks == 1) ? "" : "s",n_passes);
  dt_sum = 0.0;
  dt_min = +1.0e+100;
  dt_max = -1.0e+100;
  for(pass = -2;pass < n_passes;pass++)
  {
    (void)get_delta_time();
    //
    // first measurement, generate random data
    //
    srand(pass + 3);
    for(n = 0;n < n_blocks;n++)
      for(i = 0;i < 16;i++)
        data[i] = ((unsigned int)rand() & 0xFFFF) | (((unsigned int)rand() & 0xFFFF) << 16);
    dt1 = get_delta_time();
    //
    // second measurement, generate random data and compute the MD5 message digests
    //
    srand(pass + 3);
    for(n = 0;n < n_blocks;n++)
    {
      for(i = 0;i < 16;i++)
        data[i] = ((unsigned int)rand() & 0xFFFF) | (((unsigned int)rand() & 0xFFFF) << 16);
      md5_cpu(data,hash);
    }
    dt2 = get_delta_time();
    //
    // report
    //
    dt = dt2 - dt1;
    printf("  pass %2d: %.6f %.6f %.6f\n",pass,dt1,dt2,dt);
    dt_sum += dt;
    if(dt < dt_min)
      dt_min = dt;
    if(dt > dt_max)
      dt_max = dt;
  }
  dt_sum -= dt_min + dt_max; // ignore the smallest and largest values (discard one outlier on each side)
  printf("  %.0f MD5 message digests per second (one thread, no SIMD instructions)\n",(double)n_passes * (double)n_blocks / dt_sum);
}


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
// CUDA stuff
//


// grid 2D block 1D
__global__ void computeMD5OnGPU(unsigned int dev_data[], unsigned int dev_hash[])
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * gridDim.x * blockDim.x + ix;
# define T(i)  K  // in our unmodified MD5 message digest the T[i] constants can all be stored in the same place
  unsigned int K; // place where the T[i] constants will be stored

    if (idx < (gridDim.x * gridDim.y * blockDim.x * blockDim.y))
       MD5(((unsigned int *) dev_data+16*idx),1,((unsigned int *)dev_hash+4*idx),1);
# undef T
}

static void md5_cuda_speed(int n_blocks,int n_passes)
{
    printf("Starting md5 GPU...\n");

  // set up device
  int dev = 0;
  hipDeviceProp_t deviceProp;

  CHECK(hipGetDeviceProperties(&deviceProp, dev));
  printf("Using Device %d: %s\n", dev, deviceProp.name);
  CHECK(hipSetDevice(dev));

  // define data size
  size_t data_size;
  size_t hash_size;
  
  n_blocks &= 0xFFFF0000; // make sure that the number of blocks is a multiple of 65536
                          // (to avoid complications in the kernel launch grid)
  if(n_blocks <= 0)
    n_blocks = 1 << 24;
  if(n_blocks > (1 << 26))
    n_blocks = 1 << 26;
  for(;;)
  { // this could be done in a more elegant way ...
    data_size = (size_t)n_blocks * (size_t)16 * sizeof(unsigned int);
    hash_size = (size_t)n_blocks * (size_t) 4 * sizeof(unsigned int);
    if(data_size + hash_size <= max_device_memory)
      break;
    n_blocks -= 65536;
  }
  if(n_blocks == 0)
  {
    fprintf(stderr,"Not enough memory in the cuda device\n");
    exit(1);
  }
  if(n_passes <= 0)
    n_passes = 10;
  if(n_passes > 100)
    n_passes = 100;

  // create host and device memory areas
  //
  unsigned int *host_data;
  unsigned int *host_hash;
  unsigned int *dev_data;
  unsigned int *dev_hash;
  
  host_data = (unsigned int *)malloc(data_size);
  host_hash = (unsigned int *)malloc(hash_size);
  if(host_data == NULL || host_hash == NULL)
  {
    fprintf(stderr,"Out of host memory!\n");
    exit(1);
  }
  CHECK(hipMalloc((void **)&dev_data, data_size));
  CHECK(hipMalloc((void **)&dev_hash, hash_size));
  
  // do the hard work
  double dt,dt_sum,dt_min,dt_max;
  int i,n,pass;

  printf("md5_cuda_speed (%d block%s, %d+2 passes, %d+%d bytes per thread):\n",
         n_blocks,(n_blocks == 1) ? "" : "s",
         n_passes,
         (int)(data_size / n_blocks),
         (int)(hash_size / n_blocks));
  dt_sum = 0.0;
  dt_min = +1.0e+100;
  dt_max = -1.0e+100;

  // define configuration
  // to allow a large launch grid, we will use a 32768x(n_blocks/32768) grid
  int dimx = 512;
  int dimy = 128;
  dim3 block(64,1);
  dim3 grid(dimx, dimy);
  

  printf("execution configuration: grid(%d, %d) - block(%d, %d)\n", grid.x, grid.y, block.x, block.y);
  for(pass = -2;pass < n_passes;pass++)
  {
    // initialize the host_data and send it to the device (this data layout is not the best for the device)
   srand(pass + 3);
    for(n = 0;n < 16u * n_blocks;n++)
      host_data[n] = ((unsigned int)rand() & 0xFFFF) | (((unsigned int)rand() & 0xFFFF) << 16);
    CHECK(hipMemcpy(dev_data, host_data, data_size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_hash, host_hash, hash_size, hipMemcpyHostToDevice));

    // invoke kernel at host side(set up launch configuration first)
    (void)get_delta_time();
    computeMD5OnGPU<<<grid, block>>>(dev_data, dev_hash);
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());
    dt = get_delta_time();

    // read and check the device hash data
    CHECK(hipMemcpy(host_hash, dev_hash, hash_size, hipMemcpyDeviceToHost));
    for(i = 0;i < n_blocks;i++)
    {
      unsigned int l_hash[4];

      md5_cpu(&host_data[16 * i],l_hash);
      if(l_hash[0] != host_hash[4 * i + 0] || l_hash[1] != host_hash[4 * i + 1] || l_hash[2] != host_hash[4 * i + 2] || l_hash[3] != host_hash[4 * i + 3])
      {
        printf("MD5 hash mismatch:\n");
        printf("  %08X %08X\n",l_hash[0],host_hash[4 * i + 0]);
        printf("  %08X %08X\n",l_hash[1],host_hash[4 * i + 1]);
        printf("  %08X %08X\n",l_hash[2],host_hash[4 * i + 2]);
        printf("  %08X %08X\n",l_hash[3],host_hash[4 * i + 3]);
        exit(1);
      }
    }
    //
    // report
    //
    printf("  pass %2d: %.6f\n",pass,dt);
    dt_sum += dt;
    if(dt < dt_min)
      dt_min = dt;
    if(dt > dt_max)
      dt_max = dt;
  }
  dt_sum -= dt_min + dt_max; // ignore the smallest and largest values (discard one outlier on each side)
  printf("  %.0f MD5 message digests per second\n",(double)n_passes * (double)n_blocks / dt_sum);
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
// Main program
//

int main(int argc,char **argv)
{
  //
  // run tests
  //
  if(argc == 2 && strcmp(argv[1],"-cpu") == 0)
  {
    md5_cpu_verification();
    md5_cpu_speed(1 << 20,10);
    return 0;
  }
  if(argc == 2 && strcmp(argv[1],"-CUDA") == 0)
  {
    md5_cuda_speed(1 << 22,10);
    return 0;
  }
  //
  // usage message
  //
  fprintf(stderr,"usage: %s -cpu     # run cpu tests\n",argv[0]);
  fprintf(stderr,"       %s -CUDA    # run CUDA tests\n",argv[0]);
  return 1;
}
